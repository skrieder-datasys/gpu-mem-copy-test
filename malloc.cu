
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = 1;

  // malloc on the device

  // loop over the loop count and copy to device
  for(int i = 0; i < loopCount; i++){
    hipMalloc((void **)&dev_a, size);
    //cudaFree(dev_a);
  }

  return 0;
}
