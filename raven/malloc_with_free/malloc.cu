
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


int main(int argc, char** argv) {

  printf("Star timer\n");
  // Start the timer
  struct timeval tim;
  gettimeofday(&tim, NULL);
  double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

  // init vars
  int malloc_size_bytes, num_mallocs;

  // not enough args throw error
  if(argc < 2){
    printf("usage: %s <int malloc_size_bytes> <int number_mallocs>\n");
  }

  // take in a command line arg to set the loop count
  if(argc > 2){
    malloc_size_bytes = atoi(argv[1]);
    num_mallocs = atoi(argv[2]);
  }

  // delcare two variables
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = malloc_size_bytes;

  // loop over num_mallocs
  for(int i = 0; i < num_mallocs; i++){
    hipMalloc((void **)&dev_a, size);
    hipFree(dev_a);
  }

  // Print timing information
  gettimeofday(&tim, NULL);
  double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("%.6lf\t", (t2-t1)); // 1000000000 = 10^9, 1000000 = 10^6
  printf("END TIMER!\n");

  return 0;
}
