
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv) {

  // init vars
  int malloc_size_bytes, num_mallocs;

  // not enough args throw error
  if(argc < 2){
    printf("usage: %s <int malloc_size_bytes> <int number_mallocs>");
  }

  // take in a command line arg to set the loop count
  if(argc > 2){
    malloc_size_bytes = atoi(argv[1]);
    num_mallocs = atoi(argv[2]);
  }

  // delcare two variables
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = malloc_size_bytes;

  // loop over num_mallocs
  for(int i = 0; i < num_mallocs; i++){
    hipMalloc((void **)&dev_a, size);
  }

  return 0;
}
