
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int host_a;
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  // malloc on the device
  hipMalloc((void **)&dev_a, size);

  // copy over to device
  hipMemcpy(dev_a, &host_a, size, hipMemcpyHostToDevice);

  // call an empty kernel
  //kernel<<<1,1>>>();

  // loop over the loop count and copy to host
  for(int i = 0; i < loopCount; i++){
    hipMemcpy(&host_a, dev_a, size, hipMemcpyDeviceToHost);
  }

  // free device memory
  hipFree(dev_a);

  // return with no errors
  return 0;
}
