
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int host_a;
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  // malloc on the device
  hipMalloc((void **)&dev_a, size);

  // loop over the loop count and copy to device
  for(int i = 0; i < loopCount; i++){
    hipMemcpy(dev_a, &host_a, size, hipMemcpyHostToDevice);
  }

  // call an empty kernel
  //kernel<<<1,1>>>();

  // free device memory
  hipFree(dev_a);

  // return with no errors
  return 0;
}
