
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }
  else{
    printf("Usage: %s <# of copies>\n", argv[0]);
  }

  // delcare two variables
  int host_a;
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // malloc on the device
  hipMalloc((void **)&dev_a, size);

  // loop over the loop count and copy to device
  for(int i = 0; i < loopCount; i++){
    hipMemcpyAsync(dev_a, &host_a, size, hipMemcpyHostToDevice, stream);
  }

  hipError_t e = hipStreamSynchronize(stream);
  if( e!=hipSuccess)printf("%s\n", hipGetErrorString(e)); 

  // free device memory
  hipFree(dev_a);

  // return with no errors
  return 0;
}
