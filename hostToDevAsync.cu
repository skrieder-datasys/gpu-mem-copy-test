
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int host_a;
  int *dev_a;

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // malloc on the device
  hipMalloc((void **)&dev_a, size);

  // loop over the loop count and copy to device
  for(int i = 0; i < loopCount; i++){
    hipError_t e = hipMemcpyAsync(dev_a, &host_a, size, hipMemcpyHostToDevice, stream);
    //if( e!=cudaSuccess)printf("%s\n", cudaGetErrorString(e)); 
  }
  hipDeviceSynchronize();

  // free device memory
  hipFree(dev_a);

  // return with no errors
  return 0;
}
